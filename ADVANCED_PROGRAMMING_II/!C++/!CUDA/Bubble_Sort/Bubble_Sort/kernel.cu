#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <random>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 5

__global__ void Bubble_Sort_GPU(__int32 * array_GPU, const size_t * const size);


int main(int argc, char* argv[])
{
	//CPU VARIABLES
	constexpr size_t size = 10;
	__int32* array_to_sort = new __int32[size];

	//GPU VARIABLES
	size_t* size_GPU{};
	hipMalloc(&size_GPU, sizeof(__int32));
	hipMemcpy(size_GPU, &size, sizeof(__int32), HostToDevice);
	__int32* array_to_sort_GPU{};
	hipMalloc(&array_to_sort_GPU, sizeof(__int32) * (size));


	std::default_random_engine generator;
	std::uniform_int_distribution<> dis(-100, 100);			//zakres naszego losowania


	//FILL CPU ARRAY AND THEN COPY INTO GPU ARRAY
	for (size_t i = 0; i < size; ++i)
	{
		array_to_sort[i] = dis(generator);
	}
	//DISPLAY FILLED UP ARRAY
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array_to_sort[i] << " ";
	}
	_STD cout << NEW_LINE;

	hipMemcpy(array_to_sort_GPU, array_to_sort, (size)*sizeof(__int32), HostToDevice);


	Bubble_Sort_GPU << <1, size>> > (array_to_sort_GPU, 10);

	
	//COPY SORTED ARRAY BACK TO THE CPU
	hipMemcpy(array_to_sort, array_to_sort_GPU, (size) * sizeof(__int32), DeviceToHost);

	//DISPLAY SORTED ARRAY
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array_to_sort[i] << " ";
	}


	//DEALLOCATE CPU MEMORY
	delete[] array_to_sort;

	//DEALLOCATE GPU MEMORY
	hipFree(size_GPU);
	hipFree(array_to_sort_GPU);

	system("pause");
	return 0;
}

__global__ void Bubble_Sort_GPU(__int32* array_GPU, const size_t* const size)
{
	size_t id_x = threadIdx.x + blockIdx.x * blockDim.x;

	while (id_x < ((*size) - 1))
	{
		if (array_GPU[id_x] > array_GPU[id_x + 1])
		{
			__int32 temp{};
			temp = array_GPU[id_x];
			array_GPU[id_x] = array_GPU[id_x + 1];
			array_GPU[id_x + 1] = temp;
		}
	}
}
