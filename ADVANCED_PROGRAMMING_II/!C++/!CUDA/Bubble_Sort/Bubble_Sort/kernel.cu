#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <random>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 10

__global__ void Bubble_Sort_GPU(__int32 * array_GPU, const size_t * const size);


int main(int argc, char* argv[])
{
	//CPU VARIABLES
	constexpr size_t size = 100;
	__int32* array_to_sort = new __int32[size];

	//GPU VARIABLES
	size_t* size_GPU{};
	hipMalloc(&size_GPU, sizeof(__int32));
	hipMemcpy(size_GPU, &size, sizeof(__int32), HostToDevice);
	__int32* array_to_sort_GPU{};
	hipMalloc(&array_to_sort_GPU, sizeof(__int32) * (size));


	std::default_random_engine generator;
	std::uniform_int_distribution<> dis(-100, 100);			//zakres naszego losowania


	//FILL CPU ARRAY AND THEN COPY INTO GPU ARRAY
	for (size_t i = 0; i < size; ++i)
	{
		//array_to_sort[i] = dis(generator);
		array_to_sort[i] = (size - i);
	}
	//DISPLAY FILLED UP ARRAY
	_STD cout << "Before sorting: ";
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array_to_sort[i] << " ";
	}
	_STD cout << NEW_LINE;

	hipMemcpy(array_to_sort_GPU, array_to_sort, (size) * sizeof(__int32), HostToDevice);


	dim3 threads(size);
	Bubble_Sort_GPU <<<1, threads>>> (array_to_sort_GPU, size_GPU);

	//COPY SORTED ARRAY BACK TO THE CPU
	hipMemcpy(array_to_sort, array_to_sort_GPU, (size) * sizeof(__int32), DeviceToHost);

	//DISPLAY SORTED ARRAY
	
	_STD cout << "After sorting: ";
	for (size_t i = 0; i < size; ++i)
	{
		_STD cout << array_to_sort[i] << " ";
	}
	_STD cout << NEW_LINE;

	//DEALLOCATE CPU MEMORY
	delete[] array_to_sort;

	//DEALLOCATE GPU MEMORY
	hipFree(size_GPU);
	hipFree(array_to_sort_GPU);

	system("pause");
	return 0;
}

__global__ void Bubble_Sort_GPU(__int32* array_GPU, const size_t* const size)
{
	int id_x = threadIdx.x;
	for (size_t i = id_x; i < (*size); ++i)
	{
		for (size_t j = 0; j < ((*size) - 1) - i; ++j)
		{
			if (array_GPU[j] > array_GPU[j + 1])
			{
				__int32 temp{};
				temp = array_GPU[j];
				array_GPU[j] = array_GPU[j + 1];
				array_GPU[j + 1] = temp;
				//array_GPU[id_x + 1] = 1;
			}
		}
		__syncthreads();
	}
}
