#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 2
#define DIM 2

//CPU FUNCTIONS
void Show_Matrix(const __int32* const* const Matrix);
void Fill_Matrix(__int32* const* const Matrix, const __int32 initial_value);

//GPU FUNCTIONS
__global__ void Show_Matrix_GPU(const __int32* const Matrix);
__global__ void Multiply_Matrices(const __int32* const Matrix_GPU_A, const __int32* const Matrix_GPU_B, __int32* const Matrix_GPU_C);

int main(int argc, char* argv[])
{
	/*
		ALLOCATING NEEDED MEMORY ON CPU (exactly in RAM)
	*/
	__int32** Matrix_CPU_A;
	Matrix_CPU_A = (__int32**)malloc(SIZE * sizeof(__int32*));
	__int32** Matrix_CPU_B;
	Matrix_CPU_B = (__int32**)malloc(SIZE * sizeof(__int32*));
	__int32** Matrix_CPU_C;
	Matrix_CPU_C = (__int32**)malloc(SIZE * sizeof(__int32*));

	for (size_t i = 0; i < SIZE; ++i)
	{
		*(Matrix_CPU_A + i) = (__int32*)malloc(SIZE * sizeof(__int32));
		*(Matrix_CPU_B + i) = (__int32*)malloc(SIZE * sizeof(__int32));
		*(Matrix_CPU_C + i) = (__int32*)malloc(SIZE * sizeof(__int32));
		//Matrix_CPU[i] = (__int32*)malloc(SIZE * sizeof(__int32));
	}
	////////////////////////////////////////////////////

	//FILL MATRIX WITH RANDOM VALUES
	Fill_Matrix(Matrix_CPU_A,5);
	Fill_Matrix(Matrix_CPU_B,5);
	Fill_Matrix(Matrix_CPU_C,0);

	Matrix_CPU_A[0][0] = 1;
	Matrix_CPU_A[0][1] = 2;
	Matrix_CPU_A[1][0] = 3;
	Matrix_CPU_A[1][1] = 4;

	Matrix_CPU_B[0][0] = 4;
	Matrix_CPU_B[0][1] = 3;
	Matrix_CPU_B[1][0] = 2;
	Matrix_CPU_B[1][1] = 1;

	Matrix_CPU_C[0][0] = 0;
	Matrix_CPU_C[0][1] = 0;
	Matrix_CPU_C[1][0] = 0;
	Matrix_CPU_C[1][1] = 0;

	//SHOW FILLED UP ARRAY
	//Show_Matrix(Matrix_CPU_A);
	//Show_Matrix(Matrix_CPU_B);
	//Show_Matrix(Matrix_CPU_C);


	/*
		ALLOCATING NEEDED MEMORY ON GPU
	*/
	__int32* Matrix_GPU_A;
	hipMalloc((void**)&Matrix_GPU_A, (SIZE * SIZE) * sizeof(__int32));	//GPU interprets 2D array as a flat array !
	__int32* Matrix_GPU_B;
	hipMalloc((void**)&Matrix_GPU_B, (SIZE * SIZE) * sizeof(__int32));
	__int32* Matrix_GPU_C;
	hipMalloc((void**)&Matrix_GPU_C, (SIZE * SIZE) * sizeof(__int32));
	////////////////////////////////////////////////////


	//COPY CPU ARRAY TO GPU
	//HANDLE_ERROR(hipMemcpy(Matrix_GPU, Matrix_CPU, (SIZE * SIZE) * sizeof(__int32), HostToDevice));	//only for statics array from HOST's!

	for (size_t i = 0; i < SIZE; ++i)
	{
		hipMemcpy(Matrix_GPU_A + i * SIZE, *(Matrix_CPU_A + i), sizeof(__int32) * SIZE, HostToDevice);
		hipMemcpy(Matrix_GPU_B + i * SIZE, *(Matrix_CPU_B + i), sizeof(__int32) * SIZE, HostToDevice);
		hipMemcpy(Matrix_GPU_C + i * SIZE, *(Matrix_CPU_C + i), sizeof(__int32) * SIZE, HostToDevice);
	}

	////////////////////////////////////////////////////
	dim3 grid(2, 1);
	//MULTIPLY THE MATRICES
	Multiply_Matrices <<<grid, 1>>> (Matrix_GPU_A, Matrix_GPU_B, Matrix_GPU_C);
	//COPY FROM GPU TO CPU
	//HANDLE_ERROR(hipMemcpy(Matrix_CPU, Matrix_GPU, (SIZE * SIZE) * sizeof(__int32), DeviceToHost));	//only for statics array

	//copying data from GPU to CPU
	for (size_t i = 0; i < SIZE; ++i)
	{
		hipMemcpy(*(Matrix_CPU_C + i), Matrix_GPU_C + i * SIZE, sizeof(__int32) * SIZE, DeviceToHost);	//for dynamic allocation, only result matrix
	}
	////////////////////////////////////////////////////


	//SHOW RESULTS
	Show_Matrix(Matrix_CPU_C);

	/*
		FREEING PREVIOUSLY ALOCATED MEMORY
	*/
	//ON CPU
	for (size_t i = 0; i < SIZE; ++i)
	{
		free(*(Matrix_CPU_A + i));
		free(*(Matrix_CPU_B + i));
		free(*(Matrix_CPU_C + i));
	}
	free(Matrix_CPU_A);
	free(Matrix_CPU_B);
	free(Matrix_CPU_C);

	//ON GPU
	hipFree(Matrix_GPU_A);
	hipFree(Matrix_GPU_B);
	hipFree(Matrix_GPU_C);
	////////////////////////////////////////////////////


	system("pause");
	return 0;
}

void Show_Matrix(const __int32* const* const Matrix)
{
	for (size_t i = 0; i < SIZE; ++i)
	{
		_STD cout << "| ";
		for (size_t j = 0; j < SIZE; ++j)
		{
			_STD cout << Matrix[i][j] << " | ";
		}
		_STD cout << NEW_LINE;
	}
	_STD cout << NEW_LINE;
}

void Fill_Matrix(__int32* const* const Matrix, const __int32 initial_value)
{
	for (size_t i = 0; i < SIZE; ++i)
	{
		for (size_t j = 0; j < SIZE; ++j)
		{
			//Matrix[i][j] = (i*SIZE)+j+initial_value;
			//Matrix[i][j] = initial_value;
		}
	}
}

__global__ void Show_Matrix_GPU(const __int32* const Matrix)
{
	int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_y = threadIdx.y + blockIdx.y * blockDim.y;
	while (id_x < SIZE)
	{
		while (id_y < SIZE)
		{
			printf("| %d ", Matrix[id_y * SIZE + id_x]);
			id_y += blockDim.y * gridDim.y;
		}
		id_x += blockDim.x * gridDim.x;
	}
	printf("\n");
}

__global__ void Multiply_Matrices(const __int32* const Matrix_GPU_A, const __int32* const Matrix_GPU_B, __int32* const Matrix_GPU_C)
{
	//int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_x = blockIdx.x;
	//int id_y = threadIdx.y + blockIdx.y * blockDim.y;
	int id_y = blockIdx.y;
	/*while (id_x < SIZE)
	{
		while (id_y < SIZE)
		{
			for (int i = 0; i < SIZE; i++)
			{
				Matrix_GPU_C[id_x * SIZE + id_y] += Matrix_GPU_A[i * SIZE + id_y] * Matrix_GPU_B[i * SIZE + id_x];
			}
			id_y += blockDim.y * gridDim.y;
		}
		id_x += blockDim.x * gridDim.x;
	}*/
	/*while (id_x < SIZE && id_y < SIZE)
	{
		printf("A[%d][%d]\n", id_y, id_x);
		id_x += blockDim.x * gridDim.x;
		id_y += blockDim.y * gridDim.y;
	}
	printf("\n");*/
	while (id_x < SIZE)
	{
		while (id_y < SIZE)
		{
			printf("A[%d][%d]\n", id_x, id_y);
			id_y += gridDim.y;
		}
		id_x += gridDim.x;
	}
	printf("\n");
}