#include "hip/hip_runtime.h"
#include "../Include/cuda_kernel.cuh"

////////////////////////////////////////////////////
//GPU

#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define SIZE 10

////////////////////////////////////////////////////
//CPU

#include <stdio.h>
#include <stdlib.h>

////////////////////////////////////////////////////


							//const pointer to const size_t
__global__ void Hello_World(const size_t* const loop_repeat)
{
	//int id_x = blockIdx.x;	//for blocks (copies of blocks)
	int id_x = blockDim.x * blockIdx.x + threadIdx.x;
	while (id_x < static_cast<int>(*loop_repeat))
	{
		printf("HELLO WORLD!%c", NEW_LINE);
		//id_x += gridDim.x;	//for blocks
		id_x += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

void My_Kernel::Kernel(const __int32 & repeat_amount)
{
	//GPU MEMORY ALLOCATION
	size_t * loop_repeat{};
	hipMalloc((void**)&loop_repeat, sizeof(size_t*));

	//COPY VALUE FROM CPU(RAM) TO GPU
	hipMemcpy(loop_repeat, &repeat_amount, sizeof(__int32), HostToDevice);

	dim3 grid_size(repeat_amount);
	dim3 block_size(repeat_amount);

	Hello_World<<<1, block_size>>>(loop_repeat);
	hipError_t final_error = hipDeviceSynchronize();	//for synchronization e.g Hello_World then printf
	if (final_error == hipSuccess)
	{
		printf("%cKernel executed successfully with code: %d !%\n", NEW_LINE, final_error);
	}
	else
	{
		printf("%cKernel executed with code error: %d !\n", NEW_LINE, final_error);
	}
}