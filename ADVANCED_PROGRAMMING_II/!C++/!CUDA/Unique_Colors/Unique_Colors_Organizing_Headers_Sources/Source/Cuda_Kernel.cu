#include "hip/hip_runtime.h"
#include "../Include/cuda_kernel.cuh"

////////////////////////////////////////////////////
//GPU

#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'
#define SIZE 10

////////////////////////////////////////////////////
//CPU

#include <stdio.h>
#include <stdlib.h>

////////////////////////////////////////////////////


//__global__ void Get_String_Length(const char* const __string, size_t* length);
__global__ void Hello_World(const size_t* const loop_repeat);	//const pointer to const size_t
__global__ void Print(const char* const __string, const size_t* const loop_repeat);	//const pointer to const size_t



///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////
/*
	FUNCTIONS BODY
*/
///////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////


__global__ void Hello_World(const size_t* const loop_repeat)
{
	//int id_x = blockIdx.x;	//for blocks (copies of blocks)
	int id_x = blockDim.x * blockIdx.x + threadIdx.x;
	while (id_x < static_cast<int>(*loop_repeat))
	{
		printf("HELLO WORLD!%c", NEW_LINE);
		//id_x += gridDim.x;	//for blocks
		id_x += blockDim.x * gridDim.x;
	}
	__syncthreads();
}

__global__ void Print(const char* const __string, const size_t* const loop_repeat)
{
	int id_x = blockDim.x * blockIdx.x + threadIdx.x;
	while (id_x < static_cast<int>(*loop_repeat))
	{
		printf("%c", __string[id_x]);
		__syncthreads();
		id_x += blockDim.x * gridDim.x;
	}
}

void My_Kernel::Kernel(const __int32& repeat_amount)
{
	//GPU MEMORY ALLOCATION
	size_t* loop_repeat{};
	hipMalloc((void**)&loop_repeat, sizeof(size_t));

	//COPY VALUE FROM CPU(RAM) TO GPU
	hipMemcpy(loop_repeat, &repeat_amount, sizeof(__int32), HostToDevice);

	dim3 grid_size(repeat_amount);
	dim3 block_size(repeat_amount);

	Hello_World << <1, block_size >> > (loop_repeat);
	hipError_t final_error = hipDeviceSynchronize();	//for synchronization e.g Hello_World then printf
	if (final_error == hipSuccess)
	{
		printf("%cKernel executed successfully with code: %d !%\n", NEW_LINE, final_error);
	}
	else
	{
		printf("%cKernel executed with code error: %d !\n", NEW_LINE, final_error);
	}

	//free(static_cast<void*>(&loop_repeat));
}

void My_Kernel::Print_String(const char * my_string)
{
	//GPU MEMORY ALLOCATION
	size_t* my_string_length{};
	hipMalloc((void**)&my_string_length, sizeof(size_t));

	size_t temp = Get_String_Length(my_string);	//get the string length
	//COPY VALUE FROM CPU(RAM) TO GPU
	hipMemcpy(my_string_length, &temp, sizeof(size_t), HostToDevice);

	char* string_GPU{};
	hipMalloc((void**)&string_GPU, (*my_string_length) * sizeof(char*));

	//COPY VALUE FROM CPU(RAM) TO GPU
	hipMemcpy(string_GPU, &my_string, (*my_string_length) * sizeof(char*), HostToDevice);


	dim3 grid_size(1);
	dim3 block_size((*my_string_length));

	Print << < grid_size, block_size >> > (string_GPU, my_string_length);

	hipError_t final_error = hipDeviceSynchronize();	//for synchronization e.g Hello_World then printf
	if (final_error == hipSuccess)
	{
		printf("%cKernel executed successfully with code: %d !%\n", NEW_LINE, final_error);
	}
	else
	{
		printf("%cKernel executed with code error: %d !\n", NEW_LINE, final_error);
	}

	//free(string_GPU);
}

const size_t My_Kernel::Get_String_Length(const char* const __string)
{
	size_t string_length{};
	while (__string[string_length] != '\0')
	{
		++(string_length);
	}
	return string_length;
}