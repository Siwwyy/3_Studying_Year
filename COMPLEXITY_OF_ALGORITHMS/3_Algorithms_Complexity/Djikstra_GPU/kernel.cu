#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <vector>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 5


//CLASS OF _Djikstra_Element, for inserting values

class _Djikstra_Element
{
private:
	//////////////////////////////////////////////////////////////////////////////
	/*
		ZMIENNE PRIVATE
	*/
	int Verticle;
	int Cost;
	int Edge;
	//////////////////////////////////
	int* Connections;
	size_t _Connections_size;
	//////////////////////////////////
	//GPU CASE
	int * Verticle_GPU;
	int * Cost_GPU;
	int * Edge_GPU;
	int* Connections_GPU;
	size_t * _Connections_size_GPU;
	//////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////
public:
	//////////////////////////////////////////////////////////////////////////////
	/*
		KONSTRUKTORY PUBLIC
	*/
	__host__ __device__  _Djikstra_Element();
	__host__ __device__  _Djikstra_Element(const int Verticle, const int Cost, const int Edge);
	__host__ __device__  _Djikstra_Element(const _Djikstra_Element& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/

	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY PUBLIC
	*/
	__host__ __device__  void set_connections_size(const size_t _Connections_size);
	__host__ __device__  void set_connection(const int value, const size_t counter);
	__host__ __device__  void set_way(const int destination, const int way_lenght);
	__host__ __device__  void set_verticle(const int verticle);
	__host__ __device__  void set_cost(const int cost);
	__host__ __device__  void set_edge(const int edge);
	__host__ __device__  void set_visited_or_not(const bool visited_or_not);
	__device__  void set_verticle_GPU(const int verticle);
	__device__  void set_edge_GPU(const int edge);
	__device__  void set_cost_GPU(const int cost);
	__device__  void print_connections_GPU() const;
	__host__  void print_connections() const;
	__host__ void Copy_Values_From_CPU_To_GPU();
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY PUBLIC
	*/
	//JEDNOARGUMENTOWE
	__host__ __device__  _Djikstra_Element& operator=(const _Djikstra_Element& Object);
	//DWUARGUMENTOWE
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY PUBLIC
	*/
	__host__ __device__  int get_verticle() const;
	__host__ __device__  int get_cost() const;
	__host__ __device__  int get_edge() const;
	__device__  int get_verticle_GPU() const;
	__device__  int get_cost_GPU() const;
	__device__  int get_edge_GPU() const;
	__host__ __device__  bool get_visited_or_not() const;
	__host__ __device__  size_t get_connections_size() const;
	__host__ __device__  int& get_connections_array(const size_t counter) const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	__host__ __device__  virtual ~_Djikstra_Element();
};

//MAIN LANUCH FUNCTION
void inserter();

/*
	GLOBAL VALUES ON CPU
*/
/////////////////////////////////////////////////////////////////////////
_Djikstra_Element* Graph;
size_t _Graph_lenght;
/////////////////////////////////////////////////////////////////////////
_Djikstra_Element* Djikstra_Matrix;
size_t _Djikstra_Matrix_lenght;
//////////////////////////////////////////////////////////////////////////////
int* Visited_Nodes;
size_t _Visited_Nodes_lenght;
//////////////////////////////////////////////////////////////////////////////
std::vector<std::pair<std::pair<int, int>, int>> Destinations;
//////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////
//SELF FUNCTIONS
void Construct(const size_t _Graph_lenght);
void Push(const int value, const int destination, const int way_lenght);
void Push_Directions(const int from, const int to, const int way_lenght);
void Get_Results();
void Find_Way(const int from, const int to, const int way_lenght);
void Copy_Values_From_CPU_To_GPU_In_Class();
//GPU FUNCTION
__global__ void Print_Graph_GPU(const _Djikstra_Element* const Graph_GPU, const size_t* const _Graph_lenght_GPU);
__global__ void Print_Djikstra_Matrix_GPU(const _Djikstra_Element* const _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU);
__global__ void Print_Visited_Nodes_GPU(const int* const Visited_Nodes_GPU, const size_t* const _Visited_Nodes_lenght_GPU);
__global__ void Minimal_Spanning_Tree_Creator(const int * the_beginning, _Djikstra_Element* Graph_GPU, const size_t* const _Graph_lenght_GPU, _Djikstra_Element* _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU, int* Visited_Nodes_GPU, const size_t* const _Visited_Nodes_lenght_GPU);
//////////////////////////////////////////////////////////////////////////////


int main(int argc, char* argv[])
{
	inserter();
	//hipDeviceProp_t prop;
	//hipGetDeviceProperties(&prop, 0);
	//printf(" --- Memory Information for device %d ---\n");
	//printf("Total global mem: %ld\n", prop.totalGlobalMem);
	//printf("Total constant Mem: %ld\n", prop.totalConstMem);
	//printf("Max mem pitch: %ld\n", prop.memPitch);
	//printf("Texture Alignment: %ld\n", prop.textureAlignment);

	//printf(" --- MP Information for device %d ---\n");
	//printf("Multiprocessor count: %d\n",
	//	prop.multiProcessorCount);
	//printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
	//printf("Registers per mp: %d\n", prop.regsPerBlock);
	//printf("Threads in warp: %d\n", prop.warpSize);
	//printf("Max threads per block: %d\n",
	//	prop.maxThreadsPerBlock);
	//printf("Max thread dimensions: (%d, %d, %d)\n",
	//	prop.maxThreadsDim[0], prop.maxThreadsDim[1],
	//	prop.maxThreadsDim[2]);
	//printf("Max grid dimensions: (%d, %d, %d)\n",
	//	prop.maxGridSize[0], prop.maxGridSize[1],
	//	prop.maxGridSize[2]);
	//printf("\n");


	system("pause");
	return 0;
}


void inserter()
{
	int m = 0;			//amount of cities
	int d = 0;			//amount of ways
	int c1 = 0;			//number of city
	int c2 = 0;			//number of city
	int p = 0;			//amount of max passengers between one course
	int s = 0;			//the beginning of way
	int e = 0;			//the end of way
	int t = 0;			//amount of max passengers to move by bus
	while (true)
	{
		std::cin >> m;
		std::cin >> d;
		//_Djikstra* Djikstra_Object = new _Djikstra(m);
		Construct(m);
		while (d > 0)
		{
			std::cin >> c1;
			std::cin >> c2;
			std::cin >> p;
			//both times cause each road is in both ways
			//Djikstra_Object->push(c1, c2, p);
			Push(c1, c2, p);
			//Djikstra_Object->push(c2, c1, p);
			Push(c2, c1, p);
			--d;
			c1 = 0;
			c2 = 0;
			p = 0;
		}
		Copy_Values_From_CPU_To_GPU_In_Class();	//copy values in Djikstra Graph from CPU to GPU
		while (true)
		{
			std::cin >> s;
			std::cin >> e;
			if (s != 0 && e != 0)
			{
				std::cin >> t;
				//Djikstra_Object->push_directions(s, e, t);
				Push_Directions(s, e, t);
			}
			else
			{
				//here call all needed functions for solve the problem cause if s and e will be equal to 0 problem will be stopped immediately
				///////////////////////////////////////////////
				//Djikstra_Object->get_results();
				Get_Results();
				///////////////////////////////////////////////
				//delete Djikstra_Object;
				exit(0);
			}
			s = 0;
			e = 0;
			t = 0;
		}
		d = 0;
		m = 0;
	}
}


////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	SELF FUNCTIONS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


void Construct(const size_t _Graph_lenght)
{
	//Initialize basically those values and arrays at the beginning
	::_Graph_lenght = _Graph_lenght;	//:: means I am invoking a global variable
	::_Djikstra_Matrix_lenght = _Graph_lenght;	//:: means I am invoking a global variable
	::_Visited_Nodes_lenght = _Graph_lenght;	//:: means I am invoking a global variable
	Graph = new _Djikstra_Element[::_Graph_lenght];
	Djikstra_Matrix = new _Djikstra_Element[_Djikstra_Matrix_lenght];
	Visited_Nodes = new int[_Visited_Nodes_lenght];
	for (size_t i = 0; i < _Graph_lenght; ++i)
	{
		Graph[i].set_connections_size(_Graph_lenght);
		Djikstra_Matrix[i].set_verticle(static_cast<int>((i + 1)));
		Djikstra_Matrix[i].set_cost(0);
		Djikstra_Matrix[i].set_edge(0);
		Visited_Nodes[i] = (-1);
	}
}

void Push(const int value, const int destination, const int way_lenght)
{
	Graph[(value - 1)].set_way(destination, way_lenght);
}

void Push_Directions(const int from, const int to, const int way_lenght)
{
	Destinations.emplace_back(_STD make_pair(std::make_pair(from, to), way_lenght));
}

void Get_Results()
{
	for (typename std::vector<std::pair<std::pair<int, int>, int>>::const_iterator vec_iterator = Destinations.begin(); vec_iterator != Destinations.end(); ++vec_iterator)
	{
		Find_Way(vec_iterator->first.first, vec_iterator->first.second, vec_iterator->second);
	}
}

void Find_Way(const int from, const int to, const int way_lenght)
{
	if (from == to)
	{
		std::cout << '0' << '\n';
	}
	else
	{
		int current_verticle = 0;
		int destination_verticle = 0;
		int from_ = 0;
		int to_ = 0;

		if (from < to)
		{
			from_ = from;
			to_ = to;
		}
		else
		{
			from_ = to;
			to_ = from;
		}
		destination_verticle = from_;
		current_verticle = to_;


		//GPU FUNCTIONS
		/////////////////////////////////////////////////////////////////////////
		_Djikstra_Element* Graph_GPU;
		size_t * _Graph_lenght_GPU;
		hipMalloc((void**)&Graph_GPU, _Graph_lenght * sizeof(_Djikstra_Element));
		hipMalloc((void**)&_Graph_lenght_GPU, sizeof(size_t));
		/////////////////////////////////////////////////////////////////////////
		_Djikstra_Element* Djikstra_Matrix_GPU;
		size_t * _Djikstra_Matrix_lenght_GPU;
		hipMalloc((void**)&Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght * sizeof(_Djikstra_Element));
		hipMalloc((void**)&_Djikstra_Matrix_lenght_GPU, sizeof(size_t));
		//////////////////////////////////////////////////////////////////////////////
		int* Visited_Nodes_GPU;
		size_t * _Visited_Nodes_lenght_GPU;
		hipMalloc((void**)&Visited_Nodes_GPU, _Visited_Nodes_lenght * sizeof(int));
		hipMalloc((void**)&_Visited_Nodes_lenght_GPU, sizeof(size_t));
		//////////////////////////////////////////////////////////////////////////////
		int * from_GPU{};
		hipMalloc((void**)&from_GPU, sizeof(int));
		hipMemset(&from_GPU, 1, sizeof(int));
		hipMemcpy(from_GPU, &from_, sizeof(int), HostToDevice);
		//////////////////////////////////////////////////////////////////////////////
		
		for (size_t i = 0; i < _Djikstra_Matrix_lenght; ++i)
		{
			Djikstra_Matrix[i].set_cost(-1);	//means its a infinity
			Djikstra_Matrix[i].set_edge(0);
			Visited_Nodes[i] = (-1);
		}



		//COPYING FROM CPU TO GPU
		//COPY GRAPH 
		hipMemcpy(Graph_GPU, Graph, _Graph_lenght * sizeof(_Djikstra_Element), HostToDevice);
		hipMemcpy(_Graph_lenght_GPU, &_Graph_lenght, sizeof(size_t), HostToDevice);		
		//////////////////////////////////////////////////////////////////////////////
		//COPY Djikstra Matrix 
		hipMemcpy(Djikstra_Matrix_GPU, Djikstra_Matrix, _Djikstra_Matrix_lenght * sizeof(_Djikstra_Element), HostToDevice);
		hipMemcpy(_Djikstra_Matrix_lenght_GPU, &_Djikstra_Matrix_lenght, sizeof(size_t), HostToDevice);	
		//////////////////////////////////////////////////////////////////////////////
		//COPY Visited Nodes 
		hipMemcpy(Visited_Nodes_GPU, Visited_Nodes, _Visited_Nodes_lenght * sizeof(_Djikstra_Element), HostToDevice);
		hipMemcpy(_Visited_Nodes_lenght_GPU, &_Visited_Nodes_lenght, sizeof(size_t), HostToDevice);
		//////////////////////////////////////////////////////////////////////////////

		//HERE PASTE ALL OF FUN BETWEEN CPU AND GPU !
		dim3 blocks(1);
		dim3 threads(512);
		//printf("\n");
		Minimal_Spanning_Tree_Creator <<<1, 1>>> (from_GPU, Graph_GPU, _Graph_lenght_GPU, Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght_GPU, Visited_Nodes_GPU, _Visited_Nodes_lenght_GPU);	//create Djikstra for this case
		//hipDeviceSynchronize();
		//hipDeviceSynchronize();
		//hipDeviceSynchronize();
		//hipDeviceSynchronize();
		//hipDeviceSynchronize();
		//printf("\n");
		//printf("\n");
		//Print_Graph_GPU<<<blocks,1>>>(Graph_GPU, _Graph_lenght_GPU);	//works
		//hipDeviceSynchronize();
		//printf("\n");
		//printf("\n");
		//Print_Djikstra_Matrix_GPU <<<blocks, 1 >>>(Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght_GPU);	//works
		//hipDeviceSynchronize();
		//printf("\n");
		//printf("\n");
		//Print_Visited_Nodes_GPU <<<blocks, 1 >>>(Visited_Nodes_GPU, _Visited_Nodes_lenght_GPU);	//works
		//hipDeviceSynchronize();
		//printf("\n");
		//printf("\n");
		//hipDeviceSynchronize();
		//printf("\n");
		//Print_Visited_Nodes_GPU << <blocks, 1 >> > (Visited_Nodes_GPU, _Visited_Nodes_lenght_GPU);	//works
		//hipDeviceSynchronize();
		//printf("\n");
		_STD cin.get();


		//COPYING FROM GPU TO CPU
		//COPY GRAPH 
		hipMemcpy(Graph, Graph_GPU , _Graph_lenght * sizeof(_Djikstra_Element), DeviceToHost);
		//////////////////////////////////////////////////////////////////////////////
		//COPY Djikstra Matrix 
		hipMemcpy(Djikstra_Matrix, Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght * sizeof(_Djikstra_Element), DeviceToHost);
		//////////////////////////////////////////////////////////////////////////////
		//COPY Visited Nodes 
		hipMemcpy(Visited_Nodes, Visited_Nodes_GPU, _Visited_Nodes_lenght * sizeof(int), DeviceToHost);
		//////////////////////////////////////////////////////////////////////////////

		_STD cout << Visited_Nodes[0] << ' ' << Visited_Nodes[1] << ' ' << Visited_Nodes[2] << ' ' <<  Visited_Nodes[3] << ' ' << Visited_Nodes[4] << ' ' << Visited_Nodes[5] << ' ' << Visited_Nodes[6] << NEW_LINE;
		_STD cout << "Road through: ";
		for (int i = ::_Djikstra_Matrix_lenght - 1; i >= 0; --i)
		{
			if (Djikstra_Matrix[i].get_verticle() == destination_verticle)
			{
				_STD cout << Djikstra_Matrix[i].get_verticle();
				break;
			}
			else if (Djikstra_Matrix[i].get_verticle() == current_verticle)
			{
				current_verticle = Djikstra_Matrix[i].get_edge();
				_STD cout << Djikstra_Matrix[i].get_verticle() << " -> ";
			}
		}
		_STD cout << NEW_LINE;
		system("pause");


		/*
			MEMORY FREE
		*/
		hipFree(Graph_GPU);
		hipFree(_Graph_lenght_GPU);

		hipFree(Djikstra_Matrix_GPU);
		hipFree(_Djikstra_Matrix_lenght_GPU);

		hipFree(Visited_Nodes_GPU);
		hipFree(_Visited_Nodes_lenght_GPU);

		hipFree(from_GPU);
	}
}

void Copy_Values_From_CPU_To_GPU_In_Class()
{
	for (size_t i = 0; i < _Graph_lenght; ++i)
	{
		Graph[i].Copy_Values_From_CPU_To_GPU();
	}
}

__global__ void Print_Graph_GPU(const _Djikstra_Element* const Graph_GPU, const size_t* const _Graph_lenght_GPU)
{
	int id_x = threadIdx.x + blockDim.x * gridDim.x;
	while (id_x < *(_Graph_lenght_GPU))
	{
		Graph_GPU[id_x].print_connections_GPU();
		id_x += blockDim.x * gridDim.x;
	}
}

__global__ void Print_Djikstra_Matrix_GPU(const _Djikstra_Element* const _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU)
{
	int id_x = blockIdx.x;
	while (id_x < *(_Djikstra_Matrix_lenght_GPU))
	{
		printf("V: %d E: %d C: %d\n", _Djikstra_Matrix_GPU[id_x].get_verticle_GPU(), _Djikstra_Matrix_GPU[id_x].get_edge_GPU(), _Djikstra_Matrix_GPU[id_x].get_cost_GPU());
		id_x += blockDim.x * gridDim.x;
	}
}

__global__  void Print_Visited_Nodes_GPU(const int* const Visited_Nodes_GPU, const size_t* const _Visited_Nodes_lenght_GPU)
{
	int id_x = threadIdx.x + blockDim.x * gridDim.x;
	while (id_x < *(_Visited_Nodes_lenght_GPU))
	{
		printf(" Visited: %d ", Visited_Nodes_GPU[id_x]);
		id_x += blockDim.x * gridDim.x;
	}
}


//GPU FUNCTION
__global__ void Minimal_Spanning_Tree_Creator(const int* the_beginning, _Djikstra_Element* Graph_GPU, const size_t* const _Graph_lenght_GPU, _Djikstra_Element* _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU, int* Visited_Nodes_GPU, const size_t* const _Visited_Nodes_lenght_GPU)
{
	//printf("%d \n", *the_beginning);
	int id_x = threadIdx.x + blockDim.x * gridDim.x;
	
//	hipDeviceSynchronize();
	//Print_Djikstra_Matrix_GPU(_Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght_GPU);
	//hipDeviceSynchronize();
	//Print_Visited_Nodes_GPU(Visited_Nodes, _Visited_Nodes_lenght);

	//int current_verticle = ((*the_beginning) - 1);	//choose the beginning (by position in array (nr index))
	//int the_smallest_cost = 999999999;
	//int the_smallest_cost_position = 0;


	while (id_x < (*_Visited_Nodes_lenght_GPU))
	{
		Visited_Nodes_GPU[id_x] = (1);
		id_x += blockDim.x * gridDim.x;
	}

	/*_Djikstra_Matrix_GPU[current_verticle].set_cost_GPU(0);
	_Djikstra_Matrix_GPU[current_verticle].set_edge_GPU((current_verticle + 1));*/
	//for (size_t i = 0; i < (*_Graph_lenght_GPU); ++i)
	//{
	//	

	//	//for (size_t j = 0; j < Graph_GPU[current_verticle].get_connections_size(); ++j)
	//	//{
	//	//	if (Graph_GPU[current_verticle].get_connections_array(j) != 0 && Visited_Nodes_GPU[j] != (j + 1))
	//	//	{
	//	//		if ((_Djikstra_Matrix_GPU[j].get_cost() == -1) || ((_Djikstra_Matrix_GPU[current_verticle].get_cost() + Graph_GPU[current_verticle].get_connections_array(j)) <= _Djikstra_Matrix_GPU[j].get_cost()))
	//	//		{
	//	//			_Djikstra_Matrix_GPU[j].set_cost(_Djikstra_Matrix_GPU[current_verticle].get_cost() + Graph_GPU[current_verticle].get_connections_array(j));
	//	//			_Djikstra_Matrix_GPU[j].set_edge((current_verticle + 1));
	//	//		}
	//	//	}
	//	//}
	//	////Seeking for the smallest element in Djikstra's Matrix
	//	//for (size_t j = 0; j < (*_Djikstra_Matrix_lenght_GPU); ++j)
	//	//{
	//	//	if ((_Djikstra_Matrix_GPU[j].get_cost() <= the_smallest_cost && _Djikstra_Matrix_GPU[j].get_cost() != (-1)) && Visited_Nodes_GPU[j] != (j + 1))
	//	//	{
	//	//		the_smallest_cost = _Djikstra_Matrix_GPU[j].get_cost();
	//	//		current_verticle = static_cast<int>(j);
	//	//	}
	//	//}
	//	//the_smallest_cost = 999999999;
	//}
}

////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	_Djikstra_Element CLASS METHOD'S BODIES
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


__host__ __device__ _Djikstra_Element::_Djikstra_Element() :
	_Connections_size(0),
	Verticle(0),
	Cost(0),
	Edge(0)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ _Djikstra_Element::_Djikstra_Element(const int Verticle, const int Cost, const int Edge) :
	_Connections_size(0),
	Verticle(Verticle),
	Cost(Cost),
	Edge(Edge)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ _Djikstra_Element::_Djikstra_Element(const _Djikstra_Element& Object) :
	_Connections_size(Object._Connections_size),
	Verticle(Object.Verticle),
	Cost(Object.Cost),
	Edge(Object.Edge)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ void _Djikstra_Element::set_connections_size(const size_t _Connections_size)
{
	if (this->_Connections_size != 0)
	{
		delete[] this->Connections;
	}
	this->_Connections_size = _Connections_size;
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ void _Djikstra_Element::set_connection(const int value, const size_t counter)
{
	this->Connections[counter] = value;
}

__host__ __device__ void _Djikstra_Element::set_way(const int destination, const int way_lenght)
{
	this->Connections[(destination - 1)] = way_lenght;
}

__host__ __device__ void _Djikstra_Element::set_verticle(const int verticle)
{
	this->Verticle = verticle;
}

__host__ __device__ void _Djikstra_Element::set_cost(const int cost)
{
	this->Cost = cost;
}

__host__ __device__ void _Djikstra_Element::set_edge(const int edge)
{
	this->Edge = edge;
}

__device__ void _Djikstra_Element::set_verticle_GPU(const int verticle)
{
	*(this->Verticle_GPU) = verticle;
}

__device__ void _Djikstra_Element::set_cost_GPU(const int cost)
{
	*(this->Cost_GPU) = cost;
}

__device__ void _Djikstra_Element::set_edge_GPU(const int edge)
{
	*(this->Edge_GPU) = edge;
}

__host__ void _Djikstra_Element::print_connections() const
{
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		printf("%d ", Connections[i]);
	}
	printf("\n");
}

__host__ void _Djikstra_Element::Copy_Values_From_CPU_To_GPU()
{
	//CudaMalloc
	hipMalloc((void**)&this->Connections_GPU, this->_Connections_size * sizeof(int));
	hipMalloc((void**)&_Connections_size_GPU, sizeof(size_t));
	hipMalloc((void**)&Verticle_GPU, sizeof(int));
	hipMalloc((void**)&Edge_GPU, sizeof(int));
	hipMalloc((void**)&Cost_GPU, sizeof(size_t));
	hipMemset(&Verticle_GPU, 0, sizeof(int));
	hipMemset(&Edge_GPU, 0, sizeof(int));
	hipMemset(&Cost_GPU, 0, sizeof(int));

	//CudaMemoryCopy from CPU to GPU
	hipMemcpy(this->Connections_GPU, this->Connections, this->_Connections_size * sizeof(int), HostToDevice);
	hipMemcpy(this->_Connections_size_GPU, &this->_Connections_size, sizeof(size_t), HostToDevice);
	hipMemcpy(Verticle_GPU, &this->Verticle, sizeof(int), HostToDevice);
	hipMemcpy(Edge_GPU, &this->Edge, sizeof(int), HostToDevice);
	hipMemcpy(Cost_GPU, &this->Cost, sizeof(int), HostToDevice);
}

__device__ void _Djikstra_Element::print_connections_GPU() const
{
	for (size_t i = 0; i < *(this->_Connections_size_GPU); ++i)
	{
		printf("%d ", Connections_GPU[i]);
	}
	printf("\n");
}

__host__ __device__ _Djikstra_Element& _Djikstra_Element::operator=(const _Djikstra_Element& Object)
{
	if (this != &Object)
	{
		this->Verticle = Object.Verticle;
		this->Cost = Object.Cost;
		this->Edge = Object.Edge;
		this->_Connections_size = Object._Connections_size;
		delete[] this->Connections;
		this->Connections = new int[this->_Connections_size];
		for (size_t i = 0; i < this->_Connections_size; ++i)
		{
			this->Connections[i] = 0;
		}
	}
	return *this;
}

__host__ __device__ int _Djikstra_Element::get_verticle() const
{
	return this->Verticle;
}

__host__ __device__ int _Djikstra_Element::get_cost() const
{
	return this->Cost;
}

__host__ __device__ int _Djikstra_Element::get_edge() const
{
	return this->Edge;
}

__device__ int _Djikstra_Element::get_verticle_GPU() const
{
	return *(this->Verticle_GPU);
}

__device__ int _Djikstra_Element::get_cost_GPU() const
{
	return *(this->Cost_GPU);
}

__device__ int _Djikstra_Element::get_edge_GPU() const
{
	return *(this->Edge_GPU);
}

__host__ __device__ size_t _Djikstra_Element::get_connections_size() const
{
	return this->_Connections_size;
}

__host__ __device__ int& _Djikstra_Element::get_connections_array(const size_t counter) const
{
	return this->Connections[counter];
}

__host__ __device__ _Djikstra_Element::~_Djikstra_Element()
{
	delete[] this->Connections;
	_Connections_size = 0;
	Edge = 0;
	Cost = 0;
	Verticle = 0;

	//UNINITIALIZE GPU MEMORY
	/*extern "C"
	{
		hipFree(this->Connections_GPU);
		hipFree(this->_Connections_size_GPU);
	}*/
}