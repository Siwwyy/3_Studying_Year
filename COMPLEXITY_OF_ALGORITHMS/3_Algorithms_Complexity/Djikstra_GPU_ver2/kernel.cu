#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <vector>
#define STOPWATCH_ON
#include "StopWatch.h"

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'

#define SIZE 5


//CLASS OF _Djikstra_Element, for inserting values

class _Djikstra_Element
{
private:
	//////////////////////////////////////////////////////////////////////////////
	/*
		ZMIENNE PRIVATE
	*/
	int Verticle;
	int Cost;
	int Edge;
	//////////////////////////////////
	int* Connections;
	size_t _Connections_size;
	//////////////////////////////////
	//GPU CASE
	int* Verticle_GPU;
	int* Cost_GPU;
	int* Edge_GPU;
	int* Connections_GPU;
	size_t* _Connections_size_GPU;
	//////////////////////////////////
	//////////////////////////////////////////////////////////////////////////////
public:
	//////////////////////////////////////////////////////////////////////////////
	/*
		KONSTRUKTORY PUBLIC
	*/
	__host__  _Djikstra_Element();
	__host__  _Djikstra_Element(const int Verticle, const int Cost, const int Edge);
	__host__  _Djikstra_Element(const _Djikstra_Element& Object);
	//////////////////////////////////////////////////////////////////////////////
	/*
		FUNKCJE PUBLIC
	*/

	//////////////////////////////////////////////////////////////////////////////
	/*
		SETTERY PUBLIC
	*/
	__host__ __device__  void set_connections_size(const size_t _Connections_size);
	__host__ __device__  void set_connection(const int value, const size_t counter);
	__host__ __device__  void set_way(const int destination, const int way_lenght);
	__host__ __device__  void set_verticle(const int verticle);
	__host__ __device__  void set_cost(const int cost);
	__host__ __device__  void set_edge(const int edge);
	__host__ __device__  void set_visited_or_not(const bool visited_or_not);
	__device__  void set_verticle_GPU(const int verticle);
	__device__  void set_edge_GPU(const int edge);
	__device__  void set_cost_GPU(const int cost);
	__device__  void print_connections_GPU() const;
	__host__  void print_connections() const;
	__host__ void Copy_Values_From_CPU_To_GPU();
	__host__ void Copy_Values_From_GPU_To_CPU();
	//////////////////////////////////////////////////////////////////////////////
	/*
		OPERATORY PUBLIC
	*/
	//JEDNOARGUMENTOWE
	__host__ __device__  _Djikstra_Element& operator=(const _Djikstra_Element& Object);
	//DWUARGUMENTOWE
	//////////////////////////////////////////////////////////////////////////////
	/*
		GETTERY PUBLIC
	*/
	__host__ __device__  int get_verticle() const;
	__host__ __device__  int get_cost() const;
	__host__ __device__  int get_edge() const;
	__host__ __device__  bool get_visited_or_not() const;
	__host__ __device__  int& get_connections_array(const size_t counter) const;
	__host__ __device__  size_t get_connections_size() const;
	__device__  size_t get_connections_size_GPU() const;
	__device__  int& get_connections_array_GPU(const size_t counter) const;
	__device__  int get_verticle_GPU() const;
	__device__  int get_cost_GPU() const;
	__device__  int get_edge_GPU() const;
	//////////////////////////////////////////////////////////////////////////////
	/*
		DESTRUKTOR
	*/
	__host__  virtual ~_Djikstra_Element();
};

//MAIN LANUCH FUNCTION
void inserter();

/*
	GLOBAL VALUES ON CPU
*/
/////////////////////////////////////////////////////////////////////////
_Djikstra_Element* Graph;
size_t _Graph_lenght;
/////////////////////////////////////////////////////////////////////////
_Djikstra_Element* Djikstra_Matrix;
size_t _Djikstra_Matrix_lenght;
//////////////////////////////////////////////////////////////////////////////
std::vector<std::pair<std::pair<int, int>, int>> Destinations;
//////////////////////////////////////////////////////////////////////////////


//////////////////////////////////////////////////////////////////////////////
//SELF FUNCTIONS
void Construct(const size_t _Graph_lenght);
void Push(const int value, const int destination, const int way_lenght);
void Push_Directions(const int from, const int to, const int way_lenght);
void Get_Results();
void Find_Way(const int from, const int to, const int way_lenght);
void Copy_Values_From_CPU_To_GPU_In_Class();
void Copy_Values_From_GPU_To_CPU_In_Class();
//GPU FUNCTION
__global__ void Minimal_Spanning_Tree_Creator(_Djikstra_Element* Graph_GPU, const size_t* const _Graph_lenght_GPU, _Djikstra_Element* _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU);
//////////////////////////////////////////////////////////////////////////////


int main(int argc, char* argv[])
{
	inserter();
	return 0;
}


void inserter()
{
	int m = 0;			//amount of cities
	int d = 0;			//amount of ways
	int c1 = 0;			//number of city
	int c2 = 0;			//number of city
	int p = 0;			//amount of max passengers between one course
	int s = 0;			//the beginning of way
	int e = 0;			//the end of way
	int t = 0;			//amount of max passengers to move by bus
	while (true)
	{
		std::cin >> m;
		std::cin >> d;
		Construct(m);
		while (d > 0)
		{
			std::cin >> c1;
			std::cin >> c2;
			std::cin >> p;
			//both times cause each road is in both ways
			Push(c1, c2, p);
			Push(c2, c1, p);
			--d;
			c1 = 0;
			c2 = 0;
			p = 0;
		}
		Copy_Values_From_CPU_To_GPU_In_Class();	//copy values in Djikstra Graph from CPU to GPU
		while (true)
		{
			std::cin >> s;
			std::cin >> e;
			if (s != 0 && e != 0)
			{
				std::cin >> t;
				Push_Directions(s, e, t);
			}
			else
			{
				//here call all needed functions for solve the problem cause if s and e will be equal to 0 problem will be stopped immediately
				///////////////////////////////////////////////
				//START_STOPWATCH
				Get_Results();
				//STOP_STOPWATCH
				///////////////////////////////////////////////
				system("pause");
				exit(0);
			}
			s = 0;
			e = 0;
			t = 0;
		}
		d = 0;
		m = 0;
	}
}


////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	SELF FUNCTIONS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


void Construct(const size_t _Graph_lenght)
{
	//Initialize basically those values and arrays at the beginning
	::_Graph_lenght = _Graph_lenght;	//:: means I am invoking a global variable
	::_Djikstra_Matrix_lenght = _Graph_lenght;	//:: means I am invoking a global variable
	Graph = new _Djikstra_Element[::_Graph_lenght];
	Djikstra_Matrix = new _Djikstra_Element[_Djikstra_Matrix_lenght];
	for (size_t i = 0; i < _Graph_lenght; ++i)
	{
		Graph[i].set_connections_size(_Graph_lenght);
		Djikstra_Matrix[i].set_verticle(static_cast<int>((i + 1)));
	}
}

void Push(const int value, const int destination, const int way_lenght)
{
	Graph[(value - 1)].set_way(destination, way_lenght);
}

void Push_Directions(const int from, const int to, const int way_lenght)
{
	Destinations.emplace_back(_STD make_pair(std::make_pair(from, to), way_lenght));
}

void Get_Results()
{
	for (typename std::vector<std::pair<std::pair<int, int>, int>>::const_iterator vec_iterator = Destinations.begin(); vec_iterator != Destinations.end(); ++vec_iterator)
	{
		Find_Way(vec_iterator->first.first, vec_iterator->first.second, vec_iterator->second);
	}
}

void Find_Way(const int from, const int to, const int way_lenght)
{
	if (from == to)
	{
		std::cout << '0' << '\n';
	}
	else
	{
		int current_verticle = 0;
		int destination_verticle = 0;
		int from_ = 0;
		int to_ = 0;

		if (from < to)
		{
			from_ = from;
			to_ = to;
		}
		else
		{
			from_ = to;
			to_ = from;
		}
		destination_verticle = from_;
		current_verticle = to_;


		//GPU FUNCTIONS
		/////////////////////////////////////////////////////////////////////////
		_Djikstra_Element* Graph_GPU;
		size_t* _Graph_lenght_GPU;
		hipMalloc((void**)&Graph_GPU, _Graph_lenght * sizeof(_Djikstra_Element));
		hipMalloc((void**)&_Graph_lenght_GPU, sizeof(size_t));
		/////////////////////////////////////////////////////////////////////////
		_Djikstra_Element* Djikstra_Matrix_GPU;
		size_t* _Djikstra_Matrix_lenght_GPU;
		hipMalloc((void**)&Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght * sizeof(_Djikstra_Element));
		hipMalloc((void**)&_Djikstra_Matrix_lenght_GPU, sizeof(size_t));
		//////////////////////////////////////////////////////////////////////////////


		//COPYING FROM CPU TO GPU
		//COPY GRAPH 
		hipMemcpy(Graph_GPU, Graph, _Graph_lenght * sizeof(_Djikstra_Element), HostToDevice);
		hipMemcpy(_Graph_lenght_GPU, &_Graph_lenght, sizeof(size_t), HostToDevice);
		//////////////////////////////////////////////////////////////////////////////
		//COPY Djikstra Matrix 
		hipMemcpy(Djikstra_Matrix_GPU, Djikstra_Matrix, _Djikstra_Matrix_lenght * sizeof(_Djikstra_Element), HostToDevice);
		hipMemcpy(_Djikstra_Matrix_lenght_GPU, &_Djikstra_Matrix_lenght, sizeof(size_t), HostToDevice);
		//////////////////////////////////////////////////////////////////////////////

		//HERE PASTE ALL OF FUN BETWEEN CPU AND GPU !
		dim3 blocks(_Djikstra_Matrix_lenght);
		dim3 threads(_Djikstra_Matrix_lenght);
		START_STOPWATCH
		Minimal_Spanning_Tree_Creator <<< blocks, threads >> > (Graph_GPU, _Graph_lenght_GPU, Djikstra_Matrix_GPU, _Djikstra_Matrix_lenght_GPU);	//create Djikstra for this case
		//hipDeviceSynchronize();
		STOP_STOPWATCH

		Copy_Values_From_GPU_To_CPU_In_Class();

		_STD cout << "Road through: ";
		for (int i = ::_Djikstra_Matrix_lenght - 1; i >= 0; --i)
		{
			if (Djikstra_Matrix[i].get_verticle() == destination_verticle)
			{
				_STD cout << Djikstra_Matrix[i].get_verticle();
				break;
			}
			else if (Djikstra_Matrix[i].get_verticle() == current_verticle)
			{
				current_verticle = Djikstra_Matrix[i].get_edge();
				_STD cout << Djikstra_Matrix[i].get_verticle() << " -> ";
			}
		}
		_STD cout << NEW_LINE;
		_STD cout << NEW_LINE;
		/*
			MEMORY FREE
		*/
		hipFree(Graph_GPU);
		hipFree(_Graph_lenght_GPU);

		hipFree(Djikstra_Matrix_GPU);
		hipFree(_Djikstra_Matrix_lenght_GPU);
	}
}

void Copy_Values_From_CPU_To_GPU_In_Class()
{
	for (size_t i = 0; i < _Graph_lenght; ++i)
	{
		Graph[i].Copy_Values_From_CPU_To_GPU();
		Djikstra_Matrix[i].Copy_Values_From_CPU_To_GPU();
	}
}

void Copy_Values_From_GPU_To_CPU_In_Class()
{
	for (size_t i = 0; i < _Graph_lenght; ++i)
	{
		Djikstra_Matrix[i].Copy_Values_From_GPU_To_CPU();
	}
}

//GPU FUNCTION
__global__ void Minimal_Spanning_Tree_Creator(_Djikstra_Element* Graph_GPU, const size_t* const _Graph_lenght_GPU, _Djikstra_Element* _Djikstra_Matrix_GPU, const size_t* const _Djikstra_Matrix_lenght_GPU)
{
	int verticle = blockIdx.x;
	int id_x = threadIdx.x;
	_Djikstra_Matrix_GPU[verticle].set_cost_GPU(999999999);

	if (Graph_GPU[verticle].get_connections_array_GPU(id_x) > 0)
	{
		if (id_x < (*_Graph_lenght_GPU))
		{
			if (_Djikstra_Matrix_GPU[verticle].get_cost_GPU() > Graph_GPU[verticle].get_connections_array_GPU(id_x))
			{
				_Djikstra_Matrix_GPU[verticle].set_cost_GPU(Graph_GPU[verticle].get_connections_array_GPU(id_x));
				_Djikstra_Matrix_GPU[verticle].set_edge_GPU((id_x + 1));
			}
		}
	}
}

////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	_Djikstra_Element CLASS METHOD'S BODIES
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


__host__ _Djikstra_Element::_Djikstra_Element() :
	_Connections_size(0),
	Verticle(0),
	Cost(0),
	Edge(0)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ _Djikstra_Element::_Djikstra_Element(const int Verticle, const int Cost, const int Edge) :
	_Connections_size(0),
	Verticle(Verticle),
	Cost(Cost),
	Edge(Edge)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ _Djikstra_Element::_Djikstra_Element(const _Djikstra_Element& Object) :
	_Connections_size(Object._Connections_size),
	Verticle(Object.Verticle),
	Cost(Object.Cost),
	Edge(Object.Edge)
{
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ void _Djikstra_Element::set_connections_size(const size_t _Connections_size)
{
	if (this->_Connections_size != 0)
	{
		delete[] this->Connections;
	}
	this->_Connections_size = _Connections_size;
	this->Connections = new int[this->_Connections_size];
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		Connections[i] = 0;
	}
}

__host__ __device__ void _Djikstra_Element::set_connection(const int value, const size_t counter)
{
	this->Connections[counter] = value;
}

__host__ __device__ void _Djikstra_Element::set_way(const int destination, const int way_lenght)
{
	this->Connections[(destination - 1)] = way_lenght;
}

__host__ __device__ void _Djikstra_Element::set_verticle(const int verticle)
{
	this->Verticle = verticle;
}

__host__ __device__ void _Djikstra_Element::set_cost(const int cost)
{
	this->Cost = cost;
}

__host__ __device__ void _Djikstra_Element::set_edge(const int edge)
{
	this->Edge = edge;
}

__device__ void _Djikstra_Element::set_verticle_GPU(const int verticle)
{
	*(this->Verticle_GPU) = verticle;
}

__device__ void _Djikstra_Element::set_cost_GPU(const int cost)
{
	*(this->Cost_GPU) = cost;
}

__device__ void _Djikstra_Element::set_edge_GPU(const int edge)
{
	*(this->Edge_GPU) = edge;
}

__host__ void _Djikstra_Element::print_connections() const
{
	for (size_t i = 0; i < this->_Connections_size; ++i)
	{
		printf("%d ", Connections[i]);
	}
	printf("\n");
}

__host__ void _Djikstra_Element::Copy_Values_From_CPU_To_GPU()
{
	//CudaMalloc
	hipMalloc((void**)&this->Connections_GPU, this->_Connections_size * sizeof(int));
	hipMalloc((void**)&_Connections_size_GPU, sizeof(size_t));
	hipMalloc((void**)&Verticle_GPU, sizeof(int));
	hipMalloc((void**)&Edge_GPU, sizeof(int));
	hipMalloc((void**)&Cost_GPU, sizeof(size_t));

	//CudaMemoryCopy from CPU to GPU
	hipMemcpy(this->Connections_GPU, this->Connections, this->_Connections_size * sizeof(int), HostToDevice);
	hipMemcpy(this->_Connections_size_GPU, &this->_Connections_size, sizeof(size_t), HostToDevice);
	hipMemcpy(Verticle_GPU, &this->Verticle, sizeof(int), HostToDevice);
	hipMemcpy(Edge_GPU, &this->Edge, sizeof(int), HostToDevice);
	hipMemcpy(Cost_GPU, &this->Cost, sizeof(int), HostToDevice);
}

__host__ void _Djikstra_Element::Copy_Values_From_GPU_To_CPU()
{
	hipMemcpy(&Verticle, this->Verticle_GPU, sizeof(int), DeviceToHost);
	hipMemcpy(&Edge, this->Edge_GPU, sizeof(int), DeviceToHost);
	hipMemcpy(&Cost, this->Cost_GPU, sizeof(int), DeviceToHost);
}

__device__ void _Djikstra_Element::print_connections_GPU() const
{
	for (size_t i = 0; i < *(this->_Connections_size_GPU); ++i)
	{
		printf("%d ", Connections_GPU[i]);
	}
	printf("\n");
}

__host__ __device__ _Djikstra_Element& _Djikstra_Element::operator=(const _Djikstra_Element& Object)
{
	if (this != &Object)
	{
		this->Verticle = Object.Verticle;
		this->Cost = Object.Cost;
		this->Edge = Object.Edge;
		this->_Connections_size = Object._Connections_size;
		delete[] this->Connections;
		this->Connections = new int[this->_Connections_size];
		for (size_t i = 0; i < this->_Connections_size; ++i)
		{
			this->Connections[i] = 0;
		}
	}
	return *this;
}

__host__ __device__ int _Djikstra_Element::get_verticle() const
{
	return this->Verticle;
}

__host__ __device__ int _Djikstra_Element::get_cost() const
{
	return this->Cost;
}

__host__ __device__ int _Djikstra_Element::get_edge() const
{
	return this->Edge;
}

__device__ int _Djikstra_Element::get_verticle_GPU() const
{
	return *(this->Verticle_GPU);
}

__device__ int _Djikstra_Element::get_cost_GPU() const
{
	return *(this->Cost_GPU);
}

__device__ int _Djikstra_Element::get_edge_GPU() const
{
	return *(this->Edge_GPU);
}

__host__ __device__ size_t _Djikstra_Element::get_connections_size() const
{
	return this->_Connections_size;
}

__device__ size_t _Djikstra_Element::get_connections_size_GPU() const
{
	return *(this->_Connections_size_GPU);
}

__device__ int& _Djikstra_Element::get_connections_array_GPU(const size_t counter) const
{
	return this->Connections_GPU[counter];
}

__host__ __device__ int& _Djikstra_Element::get_connections_array(const size_t counter) const
{
	return this->Connections[counter];
}

__host__ _Djikstra_Element::~_Djikstra_Element()
{
	delete[] this->Connections;
	_Connections_size = 0;
	Edge = 0;
	Cost = 0;
	Verticle = 0;
	//UNINITIALIZE GPU MEMORY
	hipFree(Connections_GPU);
	hipFree(Verticle_GPU);
	hipFree(Edge_GPU);
	hipFree(Cost_GPU);
}