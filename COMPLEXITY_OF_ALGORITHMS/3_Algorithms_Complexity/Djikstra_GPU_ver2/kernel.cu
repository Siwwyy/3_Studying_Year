#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <vector>
#include <functional>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'


class _Djikstra_Element
{
private:
	__int32* Verticle;
	__int32* Edge;
	__int32* Cost;
public:
	__host__ _Djikstra_Element();
	__host__ _Djikstra_Element(_STD initializer_list<__int32> _Initializer);
	__host__ _Djikstra_Element(const _Djikstra_Element & Object);


	__host__ ~_Djikstra_Element();
};

//MAIN LANUCH FUNCTION
void inserter();

/*
	CPU FUNCTIONS
*/
void Print_Graph(const __int32* const* const Graph, const size_t size);
/////////////////////////////////////////////////////////////////////////

/*
	GPU FUNCTIONS
*/
__global__ void Print_Graph_GPU(const __int32* const* const Graph, const size_t * const size);
/////////////////////////////////////////////////////////////////////////


/*
	GLOBAL VALUES ON CPU
*/
/////////////////////////////////////////////////////////////////////////
__int32** Graph;	//all connections
size_t Graph_size;
/////////////////////////////////////////////////////////////////////////
__int32** Graph_GPU;
size_t* Graph_size_GPU;
/////////////////////////////////////////////////////////////////////////


int main(int argc, char* argv[])
{
	inserter();

	for (size_t i = 0; i < Graph_size; ++i)
	{
		delete[] Graph[i];
	}
	delete[] Graph;
	hipFree(Graph_GPU);
	system("pause");
	return 0;
}


void inserter()
{
	int m = 0;			//amount of cities
	int d = 0;			//amount of ways
	int c1 = 0;			//number of city
	int c2 = 0;			//number of city
	int p = 0;			//amount of max passengers between one course
	int s = 0;			//the beginning of way
	int e = 0;			//the end of way
	int t = 0;			//amount of max passengers to move by bus
	while (true)
	{
		std::cin >> m;
		std::cin >> d;
		Graph_size = m;
		Graph = new __int32*[m];
		hipMalloc((void**)&Graph_GPU, (m * m) * sizeof(__int32));	//GPU interprets 2D array as a flat array !
		hipMalloc((void**)&Graph_size_GPU, sizeof(__int32));	//GPU interprets 2D array as a flat array !
		for (size_t i = 0; i < Graph_size; ++i)
		{
			Graph[i] = new __int32[m];
			for (size_t j = 0; j < Graph_size; ++j)
			{
				Graph[i][j] = NULL;
			}
		}
		while (d > 0)
		{
			std::cin >> c1;
			std::cin >> c2;
			std::cin >> p;
			//both times cause each road is in both ways
			//Push(c1, c2, p);
			//Push(c2, c1, p);
			Graph[(c1 - 1)][(c2 - 1)] = p;
			Graph[(c2 - 1)][(c1 - 1)] = p;
			--d;
			c1 = 0;
			c2 = 0;
			p = 0;
		}

		//system("pause");
		//exit(0);
		while (true)
		{
			std::cin >> s;
			std::cin >> e;
			if (s != 0 && e != 0)
			{
				std::cin >> t;
				//Push_Directions(s, e, t);
			}
			else
			{
				//here call all needed functions for solve the problem cause if s and e will be equal to 0 problem will be stopped immediately
				///////////////////////////////////////////////
				for (size_t i = 0; i < Graph_size; ++i)
				{
					hipMemcpy(Graph_GPU + i * Graph_size, *(Graph + i), sizeof(__int32) * Graph_size, HostToDevice);
					//_STD cout << **(Graph + i) << NEW_LINE;
				}
				hipMemcpy(Graph_size_GPU, &Graph_size, sizeof(__int32), HostToDevice);
				Print_Graph(Graph, Graph_size);
				dim3 threads(Graph_size, Graph_size);
				Print_Graph_GPU<<<1, 1>>>(Graph_GPU, Graph_size_GPU);
				hipDeviceSynchronize();
				//Get_Results();
				///////////////////////////////////////////////
				//system("pause");
				//exit(0);
				break;
			}
			s = 0;
			e = 0;
			t = 0;
		}
		d = 0;
		m = 0;
		break;
	}
}

////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	CPU FUNCTIONS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


void Print_Graph(const __int32* const* const Graph, const size_t size)
{
	for (size_t i = 0; i < size; ++i)
	{
		for (size_t j = 0; j < size; ++j)
		{
			_STD cout << Graph[i][j] << ' ';
		}
		_STD cout << NEW_LINE;
	}
}

__global__ void Print_Graph_GPU(const __int32* const* const Graph, const size_t* const size)
{
	int id_x = threadIdx.x + blockIdx.x * blockDim.x;
	int id_y = threadIdx.y + blockIdx.y * blockDim.y;
//	while (id_x < *(size) && id_y < *(size))
//	{
//		printf("%d |", Graph[id_y * (*(size)) + id_x]);
///*		if (id_x % *(size) == 0 || id_y % *(size) == 0)
//		{
//			printf("\n");
//		}*/
//		id_x += blockDim.x * gridDim.x;
//		id_y += blockDim.y * gridDim.y;
//	}
	for (size_t i = 0; i < ((*size)* (*size)); ++i)
	{
		printf("%d |", Graph[i]);
	}
}


////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	_Djikstra_Element METHODS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


__host__ _Djikstra_Element::_Djikstra_Element()
{
	hipMalloc((void**)&Verticle, sizeof(__int32));
	hipMalloc((void**)&Edge, sizeof(__int32));
	hipMalloc((void**)&Cost, sizeof(__int32));
	hipMemset(Verticle, NULL, sizeof(__int32));
	hipMemset(Edge, NULL, sizeof(__int32));
	hipMemset(Cost, NULL, sizeof(__int32));
}

__host__ _Djikstra_Element::_Djikstra_Element(_STD initializer_list<__int32> _Initializer)
{
	if (_Initializer.size() == 3)
	{
		hipMalloc((void**)&Verticle, sizeof(__int32));
		hipMalloc((void**)&Edge, sizeof(__int32));
		hipMalloc((void**)&Cost, sizeof(__int32));
		hipMemset(Verticle, (*_Initializer.begin()), sizeof(__int32));
		hipMemset(Edge, (*(_Initializer.begin() + 1)), sizeof(__int32));
		hipMemset(Cost, (*(_Initializer.begin() + 2)), sizeof(__int32));
	}
}

__host__ _Djikstra_Element::~_Djikstra_Element()
{
	hipFree(Verticle);
	hipFree(Edge);
	hipFree(Cost);
}