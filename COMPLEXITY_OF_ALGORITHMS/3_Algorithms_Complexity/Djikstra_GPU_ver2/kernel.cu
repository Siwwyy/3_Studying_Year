#include "hip/hip_runtime.h"
#include ""
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"

#include <stdio.h>
#include <stdlib.h>
#include <windows.h>
#include <iostream>
#include <vector>
#include <functional>

////////////////////////////////////////////////////
//GPU

#define HostToDevice hipMemcpyHostToDevice
#define DeviceToHost hipMemcpyDeviceToHost
#define OK hipSuccess
#define NEW_LINE '\n'


class _Djikstra_Element
{
private:
	__int32* Verticle;
	__int32* Edge;
	__int32* Cost;
public:
	__host__ _Djikstra_Element();
	__host__ _Djikstra_Element(_STD initializer_list<__int32> _Initializer);
	__host__ _Djikstra_Element(const _Djikstra_Element & Object);


	__host__ ~_Djikstra_Element();
};

//MAIN LANUCH FUNCTION
void inserter();

/*
	CPU FUNCTIONS
*/
void Print_Graph(const __int32* const* const Graph, size_t size);
/////////////////////////////////////////////////////////////////////////




/*
	GLOBAL VALUES ON CPU
*/
/////////////////////////////////////////////////////////////////////////
__int32** Graph;	//all connections
size_t Graph_size;
/////////////////////////////////////////////////////////////////////////

/////////////////////////////////////////////////////////////////////////


int main(int argc, char* argv[])
{
	inserter();

	for (size_t i = 0; i < Graph_size; ++i)
	{
		delete[] Graph[i];
	}
	delete[] Graph;
	system("pause");
	return 0;
}


void inserter()
{
	int m = 0;			//amount of cities
	int d = 0;			//amount of ways
	int c1 = 0;			//number of city
	int c2 = 0;			//number of city
	int p = 0;			//amount of max passengers between one course
	int s = 0;			//the beginning of way
	int e = 0;			//the end of way
	int t = 0;			//amount of max passengers to move by bus
	while (true)
	{
		std::cin >> m;
		std::cin >> d;
		Graph_size = m;
		//Construct(m);
		Graph = new __int32* [m];
		for (size_t i = 0; i < Graph_size; ++i)
		{
			Graph[i] = new __int32[m];
			for (size_t j = 0; j < Graph_size; ++j)
			{
				Graph[i][j] = NULL;
			}
		}
		while (d > 0)
		{
			std::cin >> c1;
			std::cin >> c2;
			std::cin >> p;
			//both times cause each road is in both ways
			//Push(c1, c2, p);
			//Push(c2, c1, p);
			Graph[(c1 - 1)][(c2 - 1)] = p;
			Graph[(c2 - 1)][(c1 - 1)] = p;
			--d;
			c1 = 0;
			c2 = 0;
			p = 0;
		}
		Print_Graph(Graph, Graph_size);
		//system("pause");
		//exit(0);
		while (true)
		{
			std::cin >> s;
			std::cin >> e;
			if (s != 0 && e != 0)
			{
				std::cin >> t;
				//Push_Directions(s, e, t);
			}
			else
			{
				//here call all needed functions for solve the problem cause if s and e will be equal to 0 problem will be stopped immediately
				///////////////////////////////////////////////
				
				//Get_Results();
				
				///////////////////////////////////////////////
				//system("pause");
				//exit(0);
				break;
			}
			s = 0;
			e = 0;
			t = 0;
		}
		d = 0;
		m = 0;
		break;
	}
}

////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	CPU FUNCTIONS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


void Print_Graph(const __int32* const* const Graph, size_t size)
{
	for (size_t i = 0; i < size; ++i)
	{
		for (size_t j = 0; j < size; ++j)
		{
			_STD cout << Graph[i][j] << ' ';
		}
		_STD cout << NEW_LINE;
	}
}


////////////////////////////////////////////////////
////////////////////////////////////////////////////
/*
	_Djikstra_Element METHODS
*/
////////////////////////////////////////////////////
////////////////////////////////////////////////////


__host__ _Djikstra_Element::_Djikstra_Element()
{
	hipMalloc((void**)&Verticle, sizeof(__int32));
	hipMalloc((void**)&Edge, sizeof(__int32));
	hipMalloc((void**)&Cost, sizeof(__int32));
}

__host__ _Djikstra_Element::_Djikstra_Element(_STD initializer_list<__int32> _Initializer)
{
	if (_Initializer.size() == 3)
	{
		hipMalloc((void**)&Verticle, sizeof(__int32));
		hipMalloc((void**)&Edge, sizeof(__int32));
		hipMalloc((void**)&Cost, sizeof(__int32));
		hipMemset(Verticle, (*_Initializer.begin()), sizeof(__int32));
		hipMemset(Edge, (*(_Initializer.begin() + 1)), sizeof(__int32));
		hipMemset(Cost, (*(_Initializer.begin() + 2)), sizeof(__int32));
	}
}

__host__ _Djikstra_Element::~_Djikstra_Element()
{
	hipFree(Verticle);
	hipFree(Edge);
	hipFree(Cost);
}